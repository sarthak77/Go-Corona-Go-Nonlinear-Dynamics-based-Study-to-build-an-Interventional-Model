#include <hip/hip_runtime.h>

#include <stdio.h>
#include <vector>
#include <set>
#include <time.h>
#include <iostream>
#include <algorithm>
#include <assert.h>

using namespace std;
/*
compile instruction : nvcc percolation_centrality.cu
run instruction : ./a.out < <input_file> > <output_file>
Note that the input would be of the following form :
First line contains 2 space separated integers N and M, denoting the count of nodes and edges
M lines follow describing the edges containing 2 space separated integers u and v, denoting
there is an edge present between u and v.
Sample structure of input :
N M
u1 v1
u2 v2
.
.
.
uM vM
Note that the percolation has been assumed as a function 1/i for the simplicity, it can be changed
as desired in line 153.
*/

#define NUM_THREADS 32
#define NUM_BLOCKS 1024

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void bfs(int V, int E, int *dColumn, int *dRow, int *Distance, int *Queue,
float *Paths, int *Dist, int *Sigma)
{
	__shared__ int arr[1];
	int *QLen = arr;
	
	int rootIndex = blockIdx.x + 1;
	int *Q = Queue + (blockIdx.x)*(V+1);
	float *dPaths = Paths + (blockIdx.x)*(V+1);
	int *dDistance = Distance + (blockIdx.x)*(V+1);

	while(rootIndex <= V)
	{
		for(int i=threadIdx.x; i<=V; i+=NUM_THREADS) dPaths[i] = 0;
		for(int i=threadIdx.x; i<=V; i+=NUM_THREADS) dDistance[i] = -1;

		if(threadIdx.x==0)
		{
			*QLen = 1;
			int root = rootIndex;
			Q[0] = root;
			dPaths[root] = 1.0f;
			dDistance[root] = 0;
		}
		__syncthreads();

		int oldQLen = 0;
		while(oldQLen < *QLen)
		{
			int id = threadIdx.x;
			int	source = Q[oldQLen++];
			int degree = dRow[source+1] - dRow[source];

			while(id < degree)
			{
				int neighbour = dColumn[dRow[source]+id];
				if(dDistance[neighbour] == -1)
				{
					dDistance[neighbour] = dDistance[source]+1;
					Q[atomicAdd(QLen, 1)] = neighbour;
				}
				if(dDistance[neighbour] == dDistance[source]+1)
					dPaths[neighbour] += dPaths[source];
					
				id += NUM_THREADS;
			}
			__syncthreads();
		}
		__syncthreads();
		
		for(int i=threadIdx.x; i<=V; i+=NUM_THREADS)
		{
			if(i == 0) continue;
			Dist[(rootIndex-1)*V+i] = dDistance[i];
			Sigma[(rootIndex-1)*V+i] = dPaths[i];
		}
		rootIndex += NUM_BLOCKS;
	}
}

__global__ void percolation_estimate(int V, int E, float *dCentrality,int *sDist,int *sSigma,float *pc)
{
	int k = blockIdx.x + 1;
	while(k<=V)
	{
		float v[NUM_THREADS]; 
		for(int i=threadIdx.x; i<=V; i+=NUM_THREADS)
		{
			if(i == 0) continue;
			if(i == k) continue;
			for(int j=1; j<=V; j++)
			{
				if(i == j) continue;
				if(k == j) continue;
				v[threadIdx.x] = 0;
				if(sDist[(k-1)*V+i]+sDist[(i-1)*V+j] == sDist[(k-1)*V+j])
				{
					v[threadIdx.x] = sSigma[(k-1)*V+i]*sSigma[(i-1)*V+j];
					v[threadIdx.x] = v[threadIdx.x]/(float)(sSigma[(k-1)*V+j]);
					v[threadIdx.x] = v[threadIdx.x]*max(0.0f,pc[k]-pc[j]);
					atomicAdd(&dCentrality[i], v[threadIdx.x]);
				}
				// printf("%d %d %d %f\n",k,i,j,v[threadIdx.x]);
			}
		}
		k += NUM_BLOCKS;
	}
}

int main()
{
	int V, E;
	cin >> V >> E;

	vector <vector <int> > graph(V+1);
	for(int i=0; i<E; ++i)
	{
		int u, v;
		cin >> u >> v;
		if(u == v) continue;
		graph[u].push_back(v);
		graph[v].push_back(u);
	}

	int *hColumn = new int[2*E];
	int *hRow	 = new int[V+2];
	float *perc  = new float[V+2];

	for(int i=1;i<=V;++i)
		perc[i] = 1.0/(float)(i);
	perc[0] = perc[V+1] = 1.0;

	for(int index=0, i=1; i<=V; ++i) 
	{
		for(int j=0;j<(int)graph[i].size();++j)
		{
			int n = graph[i][j]; 
			hColumn[index++] = n;
		}
	}
	
	// Filling row array
	long count = 0;
	for(int i=0; i<=V;)
	{
		for(int j=0;j<(int)graph.size();++j)
		{
			vector<int> v = graph[i];
			hRow[i++] = count;
			count += v.size();
		}
	}
	hRow[V+1] = count;

	float *Paths;
	int *Dist, *Sigma;
	int *dColumn, *dRow, *Distance, *Queue;

	hipMalloc((void**)&dRow,    		sizeof(int)*(V+2));
	hipMalloc((void**)&Dist,			sizeof(int)*(V*V+2));
	hipMalloc((void**)&Sigma,			sizeof(int)*(V*V+2));
	hipMalloc((void**)&dColumn, 		sizeof(int)*(2*E));
	hipMalloc((void**)&Queue,    		sizeof(int)*(V+1)*NUM_BLOCKS);
	hipMalloc((void**)&Distance,		sizeof(int)*(V+1)*NUM_BLOCKS);
	hipMalloc((void**)&Paths,			sizeof(float)*(V+1)*NUM_BLOCKS);

	hipMemcpy(dRow, hRow, sizeof(int)*(V+2),hipMemcpyHostToDevice);
	hipMemcpy(dColumn, hColumn, sizeof(int)*(2*E), hipMemcpyHostToDevice);
	gpuErrchk( hipPeekAtLastError() );

	bfs <<<NUM_BLOCKS, NUM_THREADS, 32>>> (V, E, dColumn, dRow, Distance, Queue, Paths, Dist, Sigma);
	hipDeviceSynchronize();
	gpuErrchk( hipPeekAtLastError() );
	
	hipDeviceSynchronize();

	int *GetDist = new int[V*V+2];
	int *GetSigma = new int[V*V+2];
	hipMemcpy(GetDist, Dist, sizeof(int)*(V*V+2), hipMemcpyDeviceToHost);
	hipMemcpy(GetSigma, Sigma, sizeof(int)*(V*V+2), hipMemcpyDeviceToHost);
	/*
	for(int i=1; i<=V; ++i)
	{
		for(int j=1;j<=V; ++j)
		{
			printf("%d ",GetSigma[(i-1)*V+j]);
		}
		printf("\n");
	}
	*/
	float *dCentrality, *pc;
	int *sSigma,*sDist;

	hipMalloc((void**)&dCentrality,	sizeof(float)*(V+2));
	hipMalloc((void**)&sDist,			sizeof(int)*(V*V+2));
	hipMalloc((void**)&sSigma,			sizeof(int)*(V*V+2));
	hipMalloc((void**)&pc,				sizeof(float)*(V+2));

	hipMemcpy(sDist, GetDist, sizeof(int)*(V*V+2),hipMemcpyHostToDevice);
	hipMemcpy(sSigma, GetSigma, sizeof(int)*(V*V+2),hipMemcpyHostToDevice);
	hipMemcpy(pc, perc, sizeof(float)*(V+2),hipMemcpyHostToDevice);
	gpuErrchk( hipPeekAtLastError() );

	percolation_estimate <<<NUM_BLOCKS, NUM_THREADS, 32>>> (V, E, dCentrality, sDist, sSigma, pc);
	hipDeviceSynchronize();
	gpuErrchk( hipPeekAtLastError() );
	
	hipDeviceSynchronize();

	vector<pair<float,int> > perc_pair(V+1);
	vector<float> contrib(V+1);
	perc_pair[0].first = 0;
	perc_pair[0].second = 0;
    for(int i=1;i<=V;++i)
    {
		perc_pair[i].first = perc[i];
		perc_pair[i].second = i;
    }
	sort(perc_pair.begin(),perc_pair.end());
	float carry = 0,sum_x = 0;
	for(int i=1;i<=V;++i)
	{
		contrib[perc_pair[i].second] = (float)(i-1)*perc_pair[i].first-carry;
		carry += perc_pair[i].first;
		sum_x += contrib[perc_pair[i].second];
	}
	carry = 0;
	for(int i=V;i>=1;i--)
	{
		contrib[perc_pair[i].second] += carry-(float)(V-i)*perc_pair[i].first;
		carry += perc_pair[i].first;
	}

	float *Centrality = new float[V+2];
	hipMemcpy(Centrality, dCentrality, sizeof(float)*(V+2), hipMemcpyDeviceToHost);
	
	for(int i=1; i<=V; ++i)
	{
		printf("%f\n", Centrality[i]/(sum_x-contrib[i]));
	}
	
	delete[] hRow;
	delete[] hColumn;

	hipFree(Queue);
	hipFree(dRow);
	hipFree(dColumn);
	hipFree(Distance);
	return 0;
}


void printfGraph(vector <vector<int> > &graph)
{
	printf("Graph is\n");
	for(int i=1; i<graph.size(); ++i)
	{
		printf("\n%d\t", i);
		for(int j=0;j<(int)graph[i].size();++j)
		{
			int n = graph[i][j];
			printf("%d ", n);
		}
	}
}

__global__ void dPrintGraph(int V, int E, int *dRow, int *dColumn)
{
	printf("printing from device:\nRow is\n");
	for(int i=0; i<=V+1; ++i) printf("%d ", dRow[i]);
	printf("\nCol is\n");
	for(int i=0; i<2*E; ++i) printf("%d ", dColumn[i]);
	printf("\n");
}

__global__ void dPrintDist(int V, int E, int *dDistance)
{
	printf("Distances \n");
	for(int i=1; i<(V+1)*V; ++i)
	{
		if(i%(V+1)==0) { ++i; printf("\n"); }
		printf("%d ", dDistance[i]);
	}
}